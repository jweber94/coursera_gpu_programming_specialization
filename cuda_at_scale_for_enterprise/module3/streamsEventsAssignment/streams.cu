#include "hip/hip_runtime.h"
// Based on code found at https://developer.download.nvidia.com/CUDA/training/StreamsAndConcurrencyWebinar.pdf
#include "streams.h"
#include <iostream>
#include <string>
// Increments all of the values in the input arrays
__global__ void kernelA1(float *dev_mem, int n, float x)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        dev_mem[i] = dev_mem[i] + x;
    }
}

//Doubles all the values in the input arrays
__global__ void kernelB1(float *dev_mem, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        dev_mem[i] = dev_mem[i] * 2;
    }
}

// Decrements all of the values in the input arrays
__global__ void kernelA2(float *dev_mem, int n, float x)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        dev_mem[i] = dev_mem[i] - x;
    }
}

//Halves all the values in the input arrays
__global__ void kernelB2(float *dev_mem, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        dev_mem[i] = dev_mem[i] / 2;
    }
}

// This will generate an array of size numElements of random integers from 0 to 255 in pageable host memory
// The host memory has to be page-locked memory or control of streams is not guaranteed
// Note that I have added an argument for the random seed, so that you can generate the same "random" values
// for multiple runs to see the result of different actions on the same set of "random" values
__host__ float *allocateHostMemory(int numElements, int seed)
{
    seed = seed != -1 ? seed : 0;
    srand(seed);
    size_t size = numElements * sizeof(float);
    float random_max = 255.0f;

    // Allocate the host pinned memory input pointer B
    float *data;
    hipHostAlloc((void**)&data, size, hipHostMallocDefault); // pinned, non-pageable memory on host for efficient data transfer

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        // Feel free to change the max value of the random input data by replacing 255 with a smaller or larger number
        data[i] = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/random_max));
    }

    return data;
}

__host__ float * allocateDeviceMemory(int numElements)
{
    // Allocate the device input vector a
    float *dev_mem = NULL;
    size_t size = numElements * sizeof(float);
    hipError_t err = hipMalloc(&dev_mem, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector memory (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return dev_mem;
}

//Synchronous copy of data from host to device using a default stream
__host__ void copyFromHostToDeviceSync(float *host_mem, float *dev_mem, int numElements, [[maybe_unused]] hipStream_t stream)
{
    size_t size = numElements * sizeof(float);
    // Copy the host input vector to the device input vectors
    printf("Copy input data from the host memory to the CUDA device\n");
    hipError_t err = hipMemcpy(dev_mem, host_mem, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

//Asynchronous copy of data from host to device using a non-default stream
__host__ void copyFromHostToDeviceAsync(float *host_mem, float *dev_mem, int numElements, hipStream_t stream)
{
    size_t size = numElements * sizeof(float);
    // Copy the host input vector to the device input vectors
    printf("Copy input data from the host memory to the CUDA device\n");
    hipError_t err = hipMemcpyAsync(dev_mem, host_mem, size, hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

//Synchronous copy of data from device to host using the default stream
__host__ void copyFromDeviceToHostSync(float *dev_mem, float *host_mem, int numElements, [[maybe_unused]] hipStream_t stream)
{
    size_t size = numElements * sizeof(float);
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    hipError_t err = hipMemcpy(host_mem, dev_mem, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

//Synchronous copy of data from device to host using a non-default stream
__host__ void copyFromDeviceToHostAsync(float *dev_mem, float *host_mem, int numElements, hipStream_t stream)
{
    size_t size = numElements * sizeof(float);
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    hipError_t err = hipMemcpyAsync(host_mem, dev_mem, size, hipMemcpyDeviceToHost, stream);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Free device global memory
__host__ void deallocateDevMemory(float *dev_mem)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipFree(dev_mem);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ std::tuple<int, int> determineThreadBlockDimensions(int num_elements)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    return {threadsPerBlock, blocksPerGrid};
}

__host__ float * runStreamsFullAsync(float *host_mem, int num_elements)
{
    // TODO you will need to update this to allow for user input to manage how kernels are executed 
    // and will probably include extra copies to/from host/device memory. The user will input
    // an integer between 0 and 255 as the seed for the random number generator. Be creative there is
    // no wrong answer as long as all kernels are run. Output all random number sequences as a comma-separated
    // list in one line and then the results also as a CSV in one line. The file should have at least 3 pairs of lines for
    // a minimum of 6 lines of results.
    
    // Prepare all streams such that all kernels and memory copies execute asynchronously
    hipStream_t stream0, stream1, stream2, stream3, stream4, stream5;
    hipStreamCreateWithFlags(&stream0,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream1,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream3,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream4,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream5,hipStreamNonBlocking);

    // Prepare device memory based on host memory
    float* d_mem = allocateDeviceMemory(num_elements);
    copyFromHostToDeviceSync(host_mem, d_mem, num_elements, stream0); // needs to be sync - if not, the kernel would run on invalid data

    // Execute 4 kernels asynchronously on independent streams
    std::tuple<int, int> threadDims = determineThreadBlockDimensions(num_elements);
    /// Kernel 1
    // Before A1 kernel ask for user input as s(0-255), which will be the seed for the random generatsor that generates the 
    // variable x which is an argument to A1. Like below do not allow A1 to run until this input is given
    //srand(s); // original code
    /// my code
    char s;
    std::cin >> s;
    srand(static_cast<int>(s));
    /// my code end
    float x = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/random_max));
    kernelA1<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream1>>>(d_mem, num_elements, x);

    /// Kernel 2
    kernelB1<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream2>>>(d_mem, num_elements);
    
    /// Kernel 3
    // After B1 runs ask for user input (0-255) and use the input value as the seed, s, to the random number generator. 
    // You will generate the random float variable x, which is the 3rd argument to the A2 kernel.
    // You will want to hold A2 from running until this value is input, either by how you set up streams or by using events
    //srand(s); // original code
    /// my code
    std::cin >> s;
    srand(static_cast<int>(s));
    x = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/random_max));
    /// my code end
    kernelA2<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream3>>>(d_mem, num_elements, x);

    /// Kernel 4
    kernelB2<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream4>>>(d_mem, num_elements);
    
    // Copy device memory back to host asynchronously
    copyFromDeviceToHostAsync(d_mem, host_mem, num_elements, stream5);
    // Wait for all streams to be completed
    hipDeviceSynchronize();
    // This might act differently on multiple GPU system without assigning devices
    deallocateDevMemory(d_mem);
    return host_mem;
}

__host__ float * runStreamsBlockingKernel2StreamsNaive(float *host_mem, int num_elements)
{
    // TODO you will need to update this to allow for user input to manage how kernels are executed 
    // and will probably include extra copies to/from host/device memory. The user will input
    // an integer between 0 and 255 as the seed for the random number generator. Be creative there is
    // no wrong answer as long as all kernels are run. Output all random number sequences as a comma-separated
    // list in one line and then the results also as a CSV in one line. The file should have at least 3 pairs of lines for
    // a minimum of 6 lines of results.

    // Prepare all streams such that all kernels and memory copies execute asynchronously
    hipStream_t stream1, stream2, stream3, stream4;
    hipStreamCreateWithFlags(&stream1,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream3,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream4,hipStreamNonBlocking);

    // Prepare device memory based on host memory
    float *d_mem = allocateDeviceMemory(num_elements);
    copyFromHostToDeviceSync(host_mem, d_mem, num_elements, stream1);
    std::tuple<int, int> threadDims = determineThreadBlockDimensions(num_elements);
    // Execute 2 pairs of kernels asynchronous with resepect to their streams
    // The order of execution can have an effect on the blocking behaviours
    /// Kernel 1
    // Before A1 kernel ask for user input as s(0-255), which will be the seed for the random generatsor that generates the 
    // variable x which is an argument to A1. Like below do not allow A1 to run until this input is given
    char s;
    std::cin >> s;
    srand(static_cast<int>(s));
    float x = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/random_max));
    kernelA1<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream2>>>(d_mem, num_elements, x);
    /// Kernel 2 in the same stream as A1
    kernelB1<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream2>>>(d_mem, num_elements);

    // After B1 runs ask for user input (0-255) and use the input value as the seed, s, to the random number generator. 
    // You will generate the random float variable x, which is the 3rd argument to the A2 kernel.
    // You will want to hold A2 from running until this value is input, either by how you set up streams or by using events
    std::cin >> s;
    srand(static_cast<int>(s));
    x = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/random_max));
    kernelA2<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream3>>>(d_mem, num_elements, x);
    /// Kernel 4
    kernelB2<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream3>>>(d_mem, num_elements);

    // Copy device memory back to host asynchronously
    copyFromDeviceToHostAsync(d_mem, host_mem, num_elements, stream4);
    deallocateDevMemory(d_mem);
    // Wait for all streams to be completed
    hipDeviceSynchronize();
    // This might act differently on multiple GPU system without assigning devices

    return host_mem;
}

__host__ float * runStreamsBlockingKernel2StreamsOptimal(float *host_mem, int num_elements)
{
    // TODO you will need to update this to allow for user input to manage how kernels are executed 
    // and will probably include extra copies to/from host/device memory. The user will input
    // an integer between 0 and 255 as the seed for the random number generator. Be creative there is
    // no wrong answer as long as all kernels are run. Output all random number sequences as a comma-separated
    // list in one line and then the results also as a CSV in one line. The file should have at least 3 pairs of lines for
    // a minimum of 6 lines of results.

   // Prepare all streams such that all kernels and memory copies execute asynchronously

    // Prepare all streams such that all kernels and memory copies execute asynchronously
    hipStream_t stream1, stream2, stream3, stream4;
    hipStreamCreateWithFlags(&stream1,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream3,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream4,hipStreamNonBlocking);

    // Prepare device memory based on host memory
    float *d_mem = allocateDeviceMemory(num_elements);
    copyFromHostToDeviceSync(host_mem, d_mem, num_elements, stream1);
    std::tuple<int, int> threadDims = determineThreadBlockDimensions(num_elements);
    // Execute 2 pairs of kernels asynchronous with resepect to their streams
    // The order of execution can have an effect on the blocking behaviours
    /// Kernel 1
    // Before A1 kernel ask for user input as s(0-255), which will be the seed for the random generatsor that generates the 
    // variable x which is an argument to A1. Like below do not allow A1 to run until this input is given
    char s;
    std::cin >> s;
    srand(static_cast<int>(s));
    float x = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/random_max));
    kernelA1<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream2>>>(d_mem, num_elements, x);
    // After B1 runs ask for user input (0-255) and use the input value as the seed, s, to the random number generator. 
    // You will generate the random float variable x, which is the 3rd argument to the A2 kernel.
    // You will want to hold A2 from running until this value is input, either by how you set up streams or by using events
    std::cin >> s;
    srand(static_cast<int>(s));
    x = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/random_max));
    /// Kernel 2
    kernelA2<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream3>>>(d_mem, num_elements, x);
    
    /// Kernel 3
    kernelB1<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream2>>>(d_mem, num_elements);
    /// Kernel 4
    kernelB2<<<std::get<0>(threadDims), std::get<0>(threadDims), 0, stream3>>>(d_mem, num_elements);

    // Copy device memory back to host asynchronously
    copyFromDeviceToHostAsync(d_mem, host_mem, num_elements, stream4);
    deallocateDevMemory(d_mem);
    // Wait for all streams to be completed
    hipDeviceSynchronize();
    // This might act differently on multiple GPU system without assigning devices

    return host_mem;
}   

__host__ void printHostMemory(float *host_mem, int num_elments)
{
    // Output results
    printf("Host memory: ");
    for(int i = 0; i < num_elments; i++)
    {
        printf("%.6f ",host_mem[i]);
    }
    printf("\n");
}

int main()
{
    int num_elements = 255; // Can be altered but keep it less than 1/2 the memory size of global memory for full concurrency
    int rand_seed = 0; // You can set this to different values for each run but default will be the same to see the effect on data

    float * host_mem = allocateHostMemory(num_elements, rand_seed);
    printHostMemory(host_mem, num_elements);
    // FIXME: Here we should write the input input to the csv
    host_mem = runStreamsFullAsync(host_mem, num_elements);
    printHostMemory(host_mem, num_elements);
    // FIXME: Here we should write the resulting to the csv
    /// Implementation Idea of the CSV writer: write a class that opens up the file on construction and closes the file on destruction and scope this execution

    host_mem = allocateHostMemory(num_elements, 0);
    printHostMemory(host_mem, num_elements);
    host_mem = runStreamsBlockingKernel2StreamsNaive(host_mem, num_elements);
    printHostMemory(host_mem, num_elements);

    host_mem = allocateHostMemory(num_elements, 0);
    printHostMemory(host_mem, num_elements);
    host_mem = runStreamsBlockingKernel2StreamsOptimal(host_mem, num_elements);
    printHostMemory(host_mem, num_elements);

    return 0;
}