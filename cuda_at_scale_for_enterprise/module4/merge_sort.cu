#include "hip/hip_runtime.h"
#include "merge_sort.h"
#include <limits.h>
#include <iostream>
#include <string>

using namespace std;

#define min(a, b) (a < b ? a : b)
// Based on https://github.com/kevin-albert/cuda-mergesort/blob/master/mergesort.cu

__host__ std::tuple<dim3, dim3, int> parseCommandLineArguments(int argc, char** argv) 
{
    int numElements = 32;
    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    for (int i = 1; i < argc; i++) {
        if (argv[i][0] == '-' && argv[i][1] && !argv[i][2]) {
            char arg = argv[i][1];
            unsigned int* toSet = 0;
            switch(arg) {
                case 'x':
                    toSet = &threadsPerBlock.x;
                    break;
                case 'y':
                    toSet = &threadsPerBlock.y;
                    break;
                case 'z':
                    toSet = &threadsPerBlock.z;
                    break;
                case 'X':
                    toSet = &blocksPerGrid.x;
                    break;
                case 'Y':
                    toSet = &blocksPerGrid.y;
                    break;
                case 'Z':
                    toSet = &blocksPerGrid.z;
                    break;
                case 'n':
                    i++;
                    numElements = stoi(argv[i]);
                    break;
            }
            if (toSet) {
                i++;
                *toSet = (unsigned int) strtol(argv[i], 0, 10);
            }
        }
    }
    return {threadsPerBlock, blocksPerGrid, numElements};
}

__host__ long *generateRandomLongArray(int numElements)
{
    //TODO generate random array of long integers of size numElements
    long *randomLongs;
    randomLongs = static_cast<long*>(malloc(numElements*sizeof(long)));
    for (int i = 0; i < numElements; i++) {
        long tmpRand = static_cast<long>(rand()); // rand() returns a random number from 0 to RAND_MAX, then we scale to the range and add the lower bound to have a valid number in the range
        randomLongs[i] = (tmpRand > RAND_MAX/2)*(-1)*tmpRand + (tmpRand <= RAND_MAX/2)*tmpRand; // make positive as well as negative values by using a simple heuristic
    }
    return randomLongs;
}

__host__ void printHostMemory(long *host_mem, int num_elments)
{
    // Output results
    for(int i = 0; i < num_elments; i++)
    {
        printf("%ld ",host_mem[i]);
    }
    printf("\n");
}

__host__ int main(int argc, char** argv) 
{

    auto[threadsPerBlock, blocksPerGrid, numElements] = parseCommandLineArguments(argc, argv);

    long *data = generateRandomLongArray(numElements);

    printf("Unsorted data: ");
    printHostMemory(data, numElements);

    mergesort(data, numElements, threadsPerBlock, blocksPerGrid);

    printf("Sorted data: ");
    printHostMemory(data, numElements);
}

__host__ void deallocateMemory(long* dataPtr, long* swpPtr, dim3* threadPtr, dim3* blocksPtr) {
    hipError_t ret = hipFree(dataPtr);
    if (ret != hipSuccess) {
        std::cerr << "Could not free memory for the data on GPU" << std::endl;
        exit(EXIT_FAILURE);
    }
    ret = hipFree(swpPtr);
    if (ret != hipSuccess) {
        std::cerr << "Could not free memory for the swp on GPU" << std::endl;
        exit(EXIT_FAILURE);
    }
    ret = hipFree(threadPtr);
    if (ret != hipSuccess) {
        std::cerr << "Could not free memory for the threads on GPU" << std::endl;
        exit(EXIT_FAILURE);
    }
    ret = hipFree(blocksPtr);
    if (ret != hipSuccess) {
        std::cerr << "Could not free memory for the blocks on GPU" << std::endl;
        exit(EXIT_FAILURE);
    }
}

__host__ std::tuple <long* ,long* ,dim3* ,dim3*> allocateMemory(long* data, int numElements, dim3 threadsPerBlock, dim3 blocksPerGrid)
{
    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    long* D_data;
    long* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    // Actually allocate the two arrays
    hipError_t ret = hipMalloc(&D_data, numElements*sizeof(long));
    if (ret != hipSuccess) {
        std::cerr << "Could not allocate memory on the GPU for D_data" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    ret = hipMalloc(&D_swp, numElements*sizeof(long));
    if (ret != hipSuccess) {
        std::cerr << "Could not allocate memory on the GPU for D_swp" << std::endl;
        exit(EXIT_FAILURE);
    }
    

    ret = hipMalloc(&D_threads, sizeof(dim3));
    if (ret != hipSuccess) {
        std::cerr << "Could not allocate memory on the GPU for D_threads" << std::endl;
        exit(EXIT_FAILURE);
    }

    ret = hipMalloc(&D_blocks, sizeof(dim3));
    if (ret != hipSuccess) {
        std::cerr << "Could not allocate memory on the GPU for D_blocks" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Copy from our input list into the first array
    ret = hipMemcpy(D_data, data, numElements*sizeof(long), hipMemcpyHostToDevice);
    if (ret != hipSuccess) {
        std::cerr << "Could not copy data from host to D_data on GPU" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Copy the thread / block info to the GPU as well
    ret = hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    if (ret != hipSuccess) {
        std::cerr << "Could not copy data from host to D_threads on GPU" << std::endl;
        exit(EXIT_FAILURE);
    }

    ret = hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);
    if (ret != hipSuccess) {
        std::cerr << "Could not copy data from host to D_blocks on GPU" << std::endl;
        exit(EXIT_FAILURE);
    }
    return {D_data, D_swp, D_threads, D_blocks};
}

__host__ void mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {
    // device memory allocation
    auto[D_data, D_swp, D_threads, D_blocks] = allocateMemory(data, size, threadsPerBlock, blocksPerGrid); // on GPU and copy the needed data to it

    long* A = D_data; // A or D_data is where the sorted array is after the algorithm has run
    long* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    // TODO Initialize timing metrics variable(s). The implementation of this is up to you
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventRecord(start,0); // enqueue start event as first event (before the first merge sort kernel has started) - from here on the time measurement is done
    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //
    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

        // Actually call the kernel
        //arguments are long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        // Switch the input / output arrays instead of copying them around
        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    // TODO calculate and print to stdout kernel execution time
    hipEventCreate(&stop);
    hipEventRecord(stop,0); // add event to the queue which is triggered after the last gpu_mergesort kernel was finished - this is the time measurement end
    hipEventSynchronize(stop); // block host until the stop event was executed on the GPU
    /// do the actual calculation of the execution time for one complete merge sort run
    hipEventElapsedTime(&elapsedTime, start,stop);
    /// std::cout << "Elapsed time is: " << elapsedTime << std::endl; // FIXME: Make sure the grader can use the output correctly - most propably it must be turned off for the grader

    /// copy result from GPU back to CPU
    hipError_t err = hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);
    if (hipSuccess != err) {
        std::cerr << "Could not copy the result back from the GPU" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Free the GPU memory
    deallocateMemory(D_data, D_swp, D_threads, D_blocks);
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    // TODO initialize 3 long variables start, middle, and end
    // middle and end do not have values set,
    // while start is set to the width of the merge sort data span * the thread index * number of slices that this kernel will sort
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        // Break from loop when the start variable is >= size of the input array
        if (start >= size) {
            break;
        }

        // Set middle to be minimum middle index (start index plus 1/2 width) and the size of the input array
        middle = min(start + (width >> 1), size);
        // Set end to the minimum of the end index (start index plus the width of the current data window) and the size of the input array
        end = min(start + width, size);
        // Perform bottom up merege given the two available arrays and the start, middle, and end variables
        gpu_bottomUpMerge(source, dest, start, middle, end);
        // Increase the start index by the width of the current data window
        start += width;
    }
}

//
// Finally, sort something gets called by gpu_mergesort() for each slice
// Note that the pseudocode below is not necessarily 100% complete you may want to review the merge sort algorithm.
//
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;

    // Create a for look that iterates between the start and end indexes
    for (long k = start; k < end; k++) {
        // if i is before the middle index and (j is the final index or the value at i <  the value at j)
        if (i < middle && (j >= end || source[i] < source[j])) {
            // set the value in the destination array at index k to the value at index i in the source array
            dest[k] = source[i];
            // increment i
            i++;
        } else {
            // set the value in the destination array at index k to the value at index j in the source array
            dest[k] = source[j];
            // increment j ///k
            j++;
        }
    }
}